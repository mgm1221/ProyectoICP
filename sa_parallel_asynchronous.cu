
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <algorithm>
#include <chrono>
#include <tuple>
#include <math.h>
#include <string.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;
using namespace std::chrono;


__device__ __constant__ char d_sigma[4];
__device__ int *d_T_table;

int sigma_size = 4;

// Calcula valor posible de la función T y los guarda en T_table_flat
void compute_T_table_flat(int* T_table_flat, int HEIGHT, int WIDTH, int MAX_L, int MAX_K) {

    for (int i = 0; i < HEIGHT * WIDTH; ++i)
        T_table_flat[i] = 0;

    T_table_flat[0 * WIDTH + MAX_K] = 1;

    for (int L = 1; L <= MAX_L; ++L) {
        for (int k = -MAX_K; k <= MAX_K; ++k) {
            int offset_k = k + MAX_K;
            int val = 0;

            if (offset_k - 1 >= 0)
                val += T_table_flat[(L - 1) * WIDTH + offset_k - 1];

            val += (sigma_size - 2) * T_table_flat[(L - 1) * WIDTH + offset_k];

            if (offset_k + 1 < WIDTH)
                val += T_table_flat[(L - 1) * WIDTH + offset_k + 1];

            T_table_flat[L * WIDTH + offset_k] = val;
        }
    }
}


// Función para recuperar valores de T_table_flat calculados previmente
__device__ int T_lookup(const int* T_table, int L, int k, int MAX_K, int WIDTH, int HEIGHT) {
    int offset_k = k + MAX_K;
    if (L < 0 || L >= HEIGHT || offset_k < 0 || offset_k >= WIDTH)
        return 0;
    return T_table[L * WIDTH + offset_k];
}


__device__ int rand_int(int max) {
     int tid = threadIdx.x + blockIdx.x * blockDim.x;

        hiprandState state;
        unsigned long seed = clock64() + tid;
        hiprand_init(seed, tid, 0, &state);

        int result = hiprand(&state) % max;

        return result;
}


__device__ void random_sequence(char* seq, int m) {

    for (int i = 0; i < m; i++) {
        seq[i] = d_sigma[rand_int(4)];
    }
}


__device__ void random_neighbor(char* neighbor, char* current, int m) {

    int pos = rand_int(4);

    for (int i = 0; i < m; ++i) {
        neighbor[i] = current[i];
    }

    char old_char = neighbor[pos];
    char new_char;
    do {
        new_char = d_sigma[rand_int(4)];
    } while (new_char == old_char);

    neighbor[pos] = new_char;

}


__device__ int hamming_distance(const char* a, const char* b, int m) {

    int dist = 0;
    for (int i = 0; i < m; i++) {
        if (a[i] != b[i]){
            dist++;
        }
    }
    return dist;
}


// Función de evaluación
__device__ int h(char* s, char* S, int MAX_K, int WIDTH, int HEIGHT, int n,
                 int m, float th, int* d_array, int* c_array, int tid){


    int near = 0;
    char* seq;
    int d = m * th;


    memset(&d_array[tid * n], 0, sizeof(int) * n);
    memset(&c_array[tid * n], 0, sizeof(int) * n);

    for (int i = 0; i < n; i++) {
        seq = &S[i * m];
        d_array[(tid * n) + i] = hamming_distance(s, seq, m);
        c_array[(tid * n) + i] = m - d_array[(tid * n) + i];

        if (d_array[(tid * n) + i] < d) {
            near++;
        }
    }

    int f = n - near;
    int GpC;
    int gi = 0;
    double sumGpC;

    double sumP;

    if (near == 0){
        GpC = 0;
    }
    else{
        sumGpC = 0;

        for (int i = 0; i < n; i++){
            if (d_array[(tid * n) + i] < d){

                gi = 1;

                for (int j = 0; j < n; j++){
                    if (i != j){
                        sumP = 0;

                        for (int c = c_array[(tid * n) + j]; c < c_array[(tid * n) + i]; c++){
                            sumP += T_lookup(d_T_table, c_array[(tid * n) + i], c, MAX_K, WIDTH, HEIGHT) / pow(4, c_array[(tid * n) + i]);
                        }
                        gi += sumP;
                    }
                }
                sumGpC += static_cast<double>(gi) / c_array[(tid * n) + i];
            }
        }
        GpC = sumGpC / near;
    }

    return (n + 1) * f + GpC;
}


__global__ void simulated_annealing(char* flat_sequences, int MAX_K, int WIDTH, int HEIGHT, int n, int m, float th, char* results, float* qualities,
                                    int* d_array,int* c_array, char* neighbors, char* best_s, 
                                    char* current_s, int num_threads, float T_init = 1000, float alpha = 0.95, int max_iter = 1000) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= num_threads) return;

    
    hiprandState state;
    hiprand_init(1234, tid, 0, &state);

    random_sequence(&current_s[tid * m], m);

    float current_cost = h(&current_s[tid * m], flat_sequences, MAX_K, WIDTH, HEIGHT , n, m, th, d_array, c_array, tid);


    for (int i = 0; i < m; ++i) {
        best_s[(tid * m) + i] = current_s[(tid * m) + i];
    }

    float best_cost = current_cost;
    float T = T_init;

    for (int iter = 0; iter < max_iter && T > 1e-3; iter++) {

        random_neighbor(&neighbors[tid * m] , current_s, m);

        float neighbor_cost = h(&neighbors[tid * m], flat_sequences, MAX_K, WIDTH, HEIGHT, n, m, th, d_array, c_array, tid);
        float delta = neighbor_cost - current_cost;

        float r = hiprand_uniform(&state);

        if (delta > 0 || (r < expf(-delta / T))) {

            for (int i = 0; i < m; ++i) {
                current_s[(tid * m) + i] = neighbors[(tid * m) + i];
            }

            current_cost = neighbor_cost;
            if (best_cost < current_cost) {

                //best_s = corrent_s
                for (int i = 0; i < m; ++i) {
                    best_s[(tid * m) + i] = current_s[(tid * m) + i];
                }

                best_cost = current_cost;
            }
        }
        
        T *= alpha;
    }

    char* seq;
    int count = 0;
    for (int i = 0; i < n; i++) {
        seq = &flat_sequences[i * m];

        if (hamming_distance(&best_s[tid * m], seq, m) >= th * m) count++;
    }

    float quality = (count / (float)n) * 100;

    for (int i = 0; i < m; ++i) {
        results[(tid * m) + i] = best_s[(tid * m) + i];

    }

    qualities[tid] = quality;

}


int main(int argc, char *argv[])
{

    char sigma[] = {'A', 'C', 'G', 'T'};

    srand(time(NULL));

    if(argc < 5){
        cout << "faltan argumentos" << endl;
        return -1;
    }

    string i_arg = argv[1];
    string str_file = argv[2];

    string th_arg = argv[3];
    float th = stof(argv[4]);

    string T_arg = argv[5];
    float T = stof(argv[6]);

    string alpha_arg = argv[7];
    float alpha = stof(argv[8]);

    string num_threads_arg = argv[9];
    int num_threads = stoi(argv[10]);

    string threads_per_block_arg = argv[11];
    int threads_per_block = stoi(argv[12]);

    auto start = high_resolution_clock::now();

    ifstream myfile(str_file);

    size_t lastSlash = str_file.find_last_of('/');
    string filename = (lastSlash == string::npos) ? str_file : str_file.substr(lastSlash + 1);

    size_t dot = filename.find('.');
    if (dot != string::npos) {
        filename = filename.substr(0, dot);
    }

    int dash1 = filename.find('-');
    int dash2 = filename.find('-', dash1 + 1);

    int n = stoi(filename.substr(0, dash1));
    int m = stoi(filename.substr(dash1 + 1, dash2 - dash1 - 1));
    int MAX_L = n;
    int MAX_K = n;
    int HEIGHT = MAX_L + 1;
    int WIDTH = 2 * MAX_K + 1;

    int* T_table_flat = new int[WIDTH * HEIGHT];
    
    char* host_flat_sequences = new char[n * m];
    string line;
    int counter = 0 ;

    while (getline(myfile, line) && counter < n) {
        if (line.size() != (m+1)) {
            cerr << "Sequence length mismatch on line " << counter << endl;
            return -1;
        }
        memcpy(&host_flat_sequences[counter * m], line.c_str(), m);
        counter++;
    }
    myfile.close();

    compute_T_table_flat(T_table_flat, HEIGHT, WIDTH, MAX_L, MAX_K);

    /*Paralelizacion*/
    int blocks_per_grid = (num_threads + threads_per_block - 1) / threads_per_block;

    // Vectores de resultados y calidad
    char* device_flat_sequences;
    char* d_results;
    float* d_quality;
    int* device_d_array ;
    int* device_c_array;
    char* d_neighbors_flat;
    char* d_best_s_flat;
    char* d_current_s_flat;

    hipMalloc(&d_T_table, sizeof(int) * HEIGHT * WIDTH);

    //reservar memoria para los resultados
    hipMalloc(&device_flat_sequences, n * m * sizeof(char));
    hipMalloc(&d_results, num_threads * sizeof(char) * m);
    hipMalloc(&d_quality, num_threads * sizeof(float));
    hipMalloc(&device_d_array, num_threads * n * sizeof(int));
    hipMalloc(&device_c_array, num_threads * n * sizeof(int));
    hipMalloc(&d_neighbors_flat, num_threads * sizeof(char) * m);
    hipMalloc(&d_best_s_flat, num_threads* m * sizeof(char));
    hipMalloc(&d_current_s_flat, num_threads* m * sizeof(char));

    hipMemcpyToSymbol(HIP_SYMBOL(d_sigma), sigma, sizeof(char) * 4);
    
    int* temp_device_T_table;
    hipMalloc(&temp_device_T_table, sizeof(int) * HEIGHT * WIDTH);
    hipMemcpy(device_flat_sequences, host_flat_sequences, n * m * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(temp_device_T_table, T_table_flat, sizeof(int) * HEIGHT * WIDTH, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_T_table), &temp_device_T_table, sizeof(int*));

    simulated_annealing<<<blocks_per_grid, threads_per_block>>>(device_flat_sequences,
                                                                MAX_K, WIDTH, HEIGHT, 
                                                                n, m, th, d_results, d_quality, 
                                                                device_d_array, device_c_array, d_neighbors_flat, 
                                                                d_best_s_flat, d_current_s_flat, num_threads,
                                                                T, alpha, (int)T);

    char* h_results = (char*)malloc(num_threads * m * sizeof(char));
    float* h_quality = (float*)malloc(num_threads * sizeof(float));
    
    hipDeviceSynchronize();
    
    // Copia de resultados
    hipMemcpy(h_results, d_results, num_threads * m * sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy(h_quality, d_quality, num_threads * sizeof(float), hipMemcpyDeviceToHost);

    int max_quality = 0;
    int best_index = 0;
    for (int i = 0; i < num_threads; i++){  

        if (max_quality < h_quality[i]){
            max_quality = h_quality[i];
            best_index = i;

        }
    }

    auto end = high_resolution_clock::now();

    cout << "Result: ";
    for (int j = 0; j < m;j++){
        printf("%c", h_results[(best_index * m) + j]);
    }

    std::chrono::duration<double> duration = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);

    cout << " Cardinality: " << h_quality[best_index] << "%" <<
    " Time taken: " << duration.count() << "s" << endl;
    
    hipFree(d_results);
    hipFree(d_quality);
    hipFree(device_flat_sequences);
    hipFree(temp_device_T_table);
    delete[] T_table_flat;
    delete[] host_flat_sequences;
    free(h_results);
    free(h_quality);

    return 0;
}
